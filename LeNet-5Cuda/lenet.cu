#include "hip/hip_runtime.h"
/*
The MIT License(MIT)
Copyright(c) 2016 Fan Wen Jie

Permission is hereby granted, free of charge, to any person obtaining a copy
of this softwareand associated documentation files(the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions :

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
// Source: https://github.com/fan-wenjie/LeNet-5

#include "lenet.h"
#include <hip/hip_runtime.h>
#include <>
#include <memory.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cstdio>


#define CUDAMALLOC_CHECK(ptr, size)									 \
{																	 \
	if (hipMalloc((void**)&ptr, size) != hipSuccess) {		     \
		fprintf(stderr, "ERROR: hipMalloc for %s failed!\n", #ptr); \
		return -1;													 \
	}																 \
}																	 \

int CudaInit()
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ERROR: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		return -1;
	}
	return 0;
}

int CudaDeInit()
{
	hipError_t cudaStatus;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ERROR: hipDeviceReset failed!");
		return -1;
	}
	return 0;
}

int LeNetCudaAlloc(LeNet5Cuda* lenet5)
{
	// Allocate GPU buffers for LeNet5 data.
	CUDAMALLOC_CHECK(lenet5->weight0_1, INPUT * LAYER1 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight2_3, LAYER2 * LAYER3 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight4_5, LAYER4 * LAYER5 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight5_6, (LAYER5 * LENGTH_FEATURE5 * LENGTH_FEATURE5) * OUTPUT * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias0_1, LAYER1 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias2_3, LAYER3 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias4_5, LAYER5 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias5_6, OUTPUT * sizeof(double));

	return 0;
}
int LeNetCudaFree(LeNet5Cuda* lenet5)
{
	hipFree(lenet5->weight0_1);
	hipFree(lenet5->weight2_3);
	hipFree(lenet5->weight4_5);
	hipFree(lenet5->weight5_6);
	hipFree(lenet5->bias0_1);
	hipFree(lenet5->bias2_3);
	hipFree(lenet5->bias4_5);
	hipFree(lenet5->bias5_6);
	return 0;
}

int FeatureCudaAlloc(FeatureCuda* feature)
{
	CUDAMALLOC_CHECK(feature->input, INPUT * LENGTH_FEATURE0 * LENGTH_FEATURE0 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer1, LAYER1 * LENGTH_FEATURE1 * LENGTH_FEATURE1 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer2, LAYER2 * LENGTH_FEATURE2 * LENGTH_FEATURE2 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer3, LAYER3 * LENGTH_FEATURE3 * LENGTH_FEATURE3 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer4, LAYER4 * LENGTH_FEATURE4 * LENGTH_FEATURE4 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer5, LAYER5 * LENGTH_FEATURE5 * LENGTH_FEATURE5 * sizeof(double));
	CUDAMALLOC_CHECK(feature->output, OUTPUT*sizeof(double));

	return 0;
}

int FeatureCudaFree(FeatureCuda* feature)
{
	hipFree(feature->input);
	hipFree(feature->layer1);
	hipFree(feature->layer2);
	hipFree(feature->layer3);
	hipFree(feature->layer4);
	hipFree(feature->layer5);
	hipFree(feature->output);
	return 0;
}
#define CUDAMEMCPY_CHECK(src, dest, bytes, type)												      \
{																								      \
	hipError_t err;																				  \
	if ((err = hipMemcpy(dest, src, bytes, type)) != hipSuccess)								      \
		fprintf(stderr, "ERROR(%i): cudaMemCpy %s from %s to %s failed!\n", err, #type, #src, #dest); \
}
int LenetCudaUpload(LeNet5* lenet, LeNet5Cuda* lenetCuda)
{
	CUDAMEMCPY_CHECK(lenet->weight0_1, lenetCuda->weight0_1, sizeof(lenet->weight0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight2_3, lenetCuda->weight2_3, sizeof(lenet->weight2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight4_5, lenetCuda->weight4_5, sizeof(lenet->weight4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight5_6, lenetCuda->weight5_6, sizeof(lenet->weight5_6), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias0_1, lenetCuda->bias0_1, sizeof(lenet->bias0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias2_3, lenetCuda->bias2_3, sizeof(lenet->bias2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias4_5, lenetCuda->bias4_5, sizeof(lenet->bias4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias5_6, lenetCuda->bias5_6, sizeof(lenet->bias5_6), hipMemcpyHostToDevice);
	return 0;
}


#define GETLENGTH(array) (sizeof(array)/sizeof(*(array)))

#define GETCOUNT(array)  (sizeof(array)/sizeof(double))

#define FOREACH(i,count) for (int i = 0; i < count; ++i)

__global__ void ForwardConvoluteKernel(const double const* input, double* output, const double const* weight, const int inputFeatures, const int inputW, const int inputH, const double const* bias)
{
	int outFeature = blockIdx.z;
	int outputFeatures = gridDim.z;
	int threadW = blockIdx.x * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.x;
	int threadH = blockIdx.y * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.y;

	double acc = 0;

	__shared__ double inTile[LENGTH_KERNEL_TILE][LENGTH_KERNEL_TILE];
	__shared__ double weightFilter[LENGTH_KERNEL][LENGTH_KERNEL];
	
	for (int inFeature = 0; inFeature < inputFeatures; inFeature++)
	{
		if ((threadW < inputW) && (threadH < inputH))
		{
			inTile[threadIdx.y][threadIdx.x] = input[inFeature * inputH * inputW +
													 threadH * inputW +
													 threadW];
		}
		if ((threadIdx.x < LENGTH_KERNEL) && (threadIdx.y < LENGTH_KERNEL))
		{
			weightFilter[threadIdx.y][threadIdx.x] = weight[inFeature * outputFeatures * LENGTH_KERNEL * LENGTH_KERNEL +
															outFeature * LENGTH_KERNEL * LENGTH_KERNEL +
															threadIdx.y * LENGTH_KERNEL +
															threadIdx.x];
		}
		__syncthreads();

		if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
			(threadH < (inputH - LENGTH_KERNEL + 1)) && (threadW < (inputW - LENGTH_KERNEL + 1)))
		{
			for (int p = 0; p < LENGTH_KERNEL; p++)
			{
				for (int q = 0; q < LENGTH_KERNEL; q++)
				{
					acc += inTile[threadIdx.y + p][threadIdx.x + q] * weightFilter[p][q];
				}
			}
		}
		__syncthreads();
	}

	if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
		(threadH < (inputH - LENGTH_KERNEL + 1)) && (threadW < (inputW - LENGTH_KERNEL + 1)))
	{
		acc += bias[outFeature];
		output[outFeature * (inputH - LENGTH_KERNEL + 1) * (inputW - LENGTH_KERNEL + 1) + threadH * (inputW - LENGTH_KERNEL + 1) + threadW] = acc * (acc > 0);
	}
}

// Similar functionality as the code in Figure 16.4 of the textbook
void ConvolutionForward(double* input, double* output, double* weight, double* bias, const int inputFeatures, const int outputFeatures, const int inputWidth, const int inputHeight)					\
{
	/*Blocks are fixed sized tiles to allow for any size of input*/
	dim3 block(LENGTH_KERNEL_TILE, LENGTH_KERNEL_TILE, 1);
	unsigned int tilesW = ceil((float)(inputWidth - LENGTH_KERNEL + 1) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	unsigned int tilesH = ceil((float)(inputHeight - LENGTH_KERNEL + 1) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	dim3 grid(tilesW, tilesH, outputFeatures);
	ForwardConvoluteKernel <<< grid, block >>> (input, output, weight, inputFeatures, inputWidth, inputHeight, bias);
}

__global__ void ReverseConvoluteKernel(const double const* input, double* output, const double const* weight, const int inputFeatures, const int inputW, const int inputH)
{
	int outFeature = blockIdx.z;
	int outputFeatures = gridDim.z;
	int threadW = blockIdx.x * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.x;
	int threadH = blockIdx.y * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.y;

	int readX = threadW - LENGTH_KERNEL + 1;
	int readY = threadH - LENGTH_KERNEL + 1;

	double acc = 0;

	__shared__ double inTile[LENGTH_KERNEL_TILE][LENGTH_KERNEL_TILE];
	__shared__ double weightFilter[LENGTH_KERNEL][LENGTH_KERNEL];

	for (int inFeature = 0; inFeature < inputFeatures; inFeature++)
	{
		if ((readX < inputW) && (readY < inputH))
		{
			if ((readX >= 0) && (readY >= 0))
			{
				inTile[threadIdx.y][threadIdx.x] = input[inFeature * inputH * inputW +
														 readY * inputW +
														 readX];
			}
			else
			{
				inTile[threadIdx.y][threadIdx.x] = 0;
			}
		}
		if ((threadIdx.x < LENGTH_KERNEL) && (threadIdx.y < LENGTH_KERNEL))
		{
			weightFilter[LENGTH_KERNEL - 1 - threadIdx.y][LENGTH_KERNEL - 1 - threadIdx.x] = weight[outFeature * inputFeatures * LENGTH_KERNEL * LENGTH_KERNEL +
																									inFeature * LENGTH_KERNEL * LENGTH_KERNEL +
																									threadIdx.y * LENGTH_KERNEL +
																									threadIdx.x];
		}
		__syncthreads();
		if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
			(threadH < (inputH + LENGTH_KERNEL - 1)) && (threadW < (inputW + LENGTH_KERNEL - 1)))
		{
			for (int p = 0; p < LENGTH_KERNEL; p++)
			{
				for (int q = 0; q < LENGTH_KERNEL; q++)
				{
					if (((threadH + p) < (inputH + LENGTH_KERNEL - 1)) && ((threadW + q) < (inputW + LENGTH_KERNEL - 1)))
						acc += inTile[threadIdx.y + p][threadIdx.x + q] * weightFilter[p][q];
				}
			}
		}
		__syncthreads();
	}

	if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
		(threadH < (inputH + LENGTH_KERNEL - 1)) && (threadW < (inputW + LENGTH_KERNEL - 1)))
	{
		output[outFeature * (inputH + LENGTH_KERNEL - 1) * (inputW + LENGTH_KERNEL - 1) + threadH * (inputW + LENGTH_KERNEL - 1) + threadW] = acc;
	}
}

__global__ void BackwardRelugrad(const double const* input, double* error, const int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < size)
	{
		error[i] *= input[i] > 0;
	}
}

__global__ void BiasUpdate(double* bias, const double const* error, const int features, const int size)
{
	double acc = 0;
	if (threadIdx.x < features)
	{
		for (int i = 0; i < size; i++)
		{
			acc += error[threadIdx.x * size + i];
		}
		bias[threadIdx.x] = acc;
	}
}

__global__ void WeightConvoluteKernel(const double const* input, double* weight, const double const* output)
{
	double acc = 0;

	extern __shared__ double data[];

	data[threadIdx.y * blockDim.x + threadIdx.x] = input[blockIdx.x * blockDim.y * blockDim.x +
														 threadIdx.y * blockDim.x +
														 threadIdx.x];
	if ((threadIdx.y < (blockDim.y - LENGTH_KERNEL + 1)) && (threadIdx.x < (blockDim.x - LENGTH_KERNEL + 1)))
	{
		data[blockDim.y * blockDim.x + threadIdx.y * (blockDim.x - LENGTH_KERNEL + 1) + threadIdx.x] = output[blockIdx.y * (blockDim.y - LENGTH_KERNEL + 1) * (blockDim.x - LENGTH_KERNEL + 1) +
																											  threadIdx.y * (blockDim.x - LENGTH_KERNEL + 1) +
																											  threadIdx.x];
	}
	__syncthreads();

	if ((threadIdx.y < LENGTH_KERNEL) && (threadIdx.x < LENGTH_KERNEL))
	{
		for (int p = 0; p < blockDim.y - LENGTH_KERNEL + 1; p++)
		{
			for (int q = 0; q < blockDim.x - LENGTH_KERNEL + 1; q++)
			{
				acc += data[(threadIdx.y + p) * blockDim.x + (threadIdx.x + q)] * data[blockDim.y * blockDim.x + p * (blockDim.x - LENGTH_KERNEL + 1) + q];
			}
		}
		weight[blockIdx.x * gridDim.y * LENGTH_KERNEL * LENGTH_KERNEL +
			   blockIdx.y * LENGTH_KERNEL * LENGTH_KERNEL +
		       threadIdx.y * LENGTH_KERNEL +
			   threadIdx.x] = acc;
	}
}

void ConvolutionBackward(double* input, double* inError, double* outError, double* weight, double* weightDeltas, double* biasDeltas,
	const int inputFeatures, const int outputFeatures, const int inputWidth, const int inputHeight)
{
	/*Blocks are fixed sized tiles to allow for any size of input*/
	dim3 block(LENGTH_KERNEL_TILE, LENGTH_KERNEL_TILE, 1);
	unsigned int tilesW = ceil((float)(inputWidth) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	unsigned int tilesH = ceil((float)(inputHeight) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	dim3 grid(tilesW, tilesH, inputFeatures);
	ReverseConvoluteKernel <<< grid, block >>> (outError, inError, weight, outputFeatures, inputWidth - LENGTH_KERNEL + 1, inputHeight - LENGTH_KERNEL + 1);

	BackwardRelugrad <<< ceil(((float)(inputFeatures * inputWidth * inputHeight)) / ((float)(LENGTH_KERNEL_TILE * LENGTH_KERNEL_TILE))),
						LENGTH_KERNEL_TILE* LENGTH_KERNEL_TILE >>> (input, inError, inputFeatures * inputWidth * inputHeight);
	BiasUpdate <<< 1, outputFeatures >>> (biasDeltas, outError, outputFeatures, (inputWidth - LENGTH_KERNEL + 1) * (inputHeight - LENGTH_KERNEL + 1));
	
	/*Kernel does not support any size feature. ran out of time to make generalized*/
	block = dim3(inputWidth, inputHeight, 1);
	grid = dim3(inputFeatures, outputFeatures, 1);
	WeightConvoluteKernel <<< grid, block, (inputWidth * inputHeight + (inputWidth - LENGTH_KERNEL + 1) * (inputHeight - LENGTH_KERNEL + 1)) * sizeof(double) >>> 
								(input, weightDeltas, outError);
}

__global__ void CUDA_SubsampForward(const double const* input, double* output, const uint32_t len, const uint32_t lenFeatIn, const uint32_t lenFeatOut)
{
	const uint32_t o1 = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t o0 = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t i = threadIdx.z + blockIdx.z * blockDim.z;

	if (o0 < lenFeatOut && o1 < lenFeatOut)
	{
		int32_t x0 = 0;
		int32_t x1 = 0;
		int32_t ismax;
		for (int32_t l0 = 0; l0 < len; ++l0)
		{
			for (int32_t l1 = 0; l1 < len; ++l1)
			{
				ismax = input[(i)*lenFeatIn * lenFeatIn + (o0 * len + l0) * lenFeatIn + (o1 * len + l1)] >
					input[(i)*lenFeatIn * lenFeatIn + (o0 * len + x0) * lenFeatIn + (o1 * len + x1)];
				x0 += ismax * (l0 - x0);
				x1 += ismax * (l1 - x1);
			}
		}
		output[(i)*lenFeatOut * lenFeatOut + (o0)*lenFeatOut + o1] =
			input[(i)*lenFeatIn * lenFeatIn + (o0 * len + x0) * lenFeatIn + (o1 * len + x1)];
	}
}
void SubsampForward(double* input, double* output, size_t insize, size_t inlayersize, size_t outsize, size_t outlayersise)
{
	size_t inTotalSize = insize * inlayersize * inlayersize;
	size_t outTotalSize = outsize * outlayersise * outlayersise;

	uint32_t len = inlayersize / outlayersise;
	{
		dim3 threads = { 16, 16, 1 };
		dim3 blocks = {
			(uint32_t)ceil((double)outlayersise / threads.x),
			(uint32_t)ceil((double)outlayersise / threads.y),
			(uint32_t)ceil((double)outsize / threads.z)
		};
		CUDA_SubsampForward << <blocks, threads >> > (input, output, len, inlayersize, outlayersise);
	}
}

__global__ void CUDA_DotBinerror(const double const* input, double* inerror, const double const* outerror, const double const* weight, double* bd, const size_t w1size, const size_t w2size)
{
	// 120x threads
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x < w1size)
	{
		double acc = 0.0;
		for (uint32_t y = 0; y < w2size; ++y)
		{
			acc += outerror[y] * weight[y + x * w2size];
		}
		inerror[x] = acc * (input[x] > 0);
	}
	if (x < w2size)
		bd[x] = outerror[x];
}
__global__ void CUDA_DotBias(const double const* input, const double const* outerror, double* wd, const size_t w1size, const size_t w2size)
{
	// 8 blocks, 16 x 16 threads
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x < w1size && y < w2size)
		wd[y + x * w2size] = input[x] * outerror[y];
}
void DotProductBackward(double* input, double* inerror, double* outerror, double* weight, double* wd, double* bd, const size_t w1size, const size_t w2size)
{
	{
		uint32_t threads = w1size;
		uint32_t blocks = 1;
		CUDA_DotBinerror << <blocks, threads >> > (input, inerror, outerror, weight, bd, w1size, w2size);
	}
	{
		// 120 x 10
		dim3 threads = { 16, 16, 1 };
		dim3 blocks = {
			(uint32_t)ceil(((double)w1size / threads.x)),
			(uint32_t)ceil(((double)w2size / threads.y)),
			1 };
		CUDA_DotBias << <blocks, threads >> > (input, outerror, wd, w1size, w2size);
	}
}

__global__ void CUDA_SubsampBackward(const double const* input, double* inerror, double* outerror, const uint32_t len, const uint32_t lenFeatIn, const uint32_t lenFeatOut)
{
	const uint32_t o1 = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t o0 = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t i = threadIdx.z + blockIdx.z * blockDim.z;

	if (o0 < lenFeatOut && o1 < lenFeatOut)
	{
		int32_t x0 = 0;
		int32_t x1 = 0;
		int32_t ismax;
		for (int32_t l0 = 0; l0 < len; ++l0)
		{
			for (int32_t l1 = 0; l1 < len; ++l1)
			{
				ismax = input[(i)*lenFeatIn * lenFeatIn + (o0 * len + l0) * lenFeatIn + (o1 * len + l1)] >
					input[(i)*lenFeatIn * lenFeatIn + (o0 * len + x0) * lenFeatIn + (o1 * len + x1)];
				x0 += ismax * (l0 - x0);
				x1 += ismax * (l1 - x1);
			}
		}
		inerror[(i)*lenFeatIn * lenFeatIn + (o0 * len + x0) * lenFeatIn + (o1 * len + x1)] =
			outerror[(i)*lenFeatOut * lenFeatOut + (o0)*lenFeatOut + (o1)];
	}
}
void SubsampBackward(double* input, double* inerror, double* outerror, size_t inlayersize, size_t outsize, size_t outlayersise)
{
	uint32_t len = inlayersize / outlayersise;
	{
		dim3 threads = { 16, 16, 1 };
		dim3 blocks = {
			(uint32_t)ceil((double)outlayersise / threads.x),
			(uint32_t)ceil((double)outlayersise / threads.y),
			(uint32_t)ceil((double)outsize / threads.z)
		};
		CUDA_SubsampBackward << <threads, blocks >> > (input, inerror, outerror, len, inlayersize, outlayersise);
	}
}

__global__ void CUDA_DotF(double* output, const double const* input, const double const* weight, const double const* bias, const size_t w1size, const size_t w2size)
{
	const uint32_t ioutput = threadIdx.x + blockIdx.x * blockDim.x;

	if (ioutput < w2size)
	{
		double acc = 0.0;
		for (uint32_t x = 0; x < w1size; ++x)
			acc += input[x] * weight[ioutput + x * w2size];
		acc += bias[ioutput];
		output[ioutput] = acc * (acc > 0);
	}
}

void DotProductForward(double* input, double* output, double* weight, size_t w1size, size_t w2size, double* bias)
{
	// 1D
	{
		int32_t threads = w2size;
		int32_t blocks = 1;

		CUDA_DotF << <blocks, threads >> > (output, input, weight, bias, w1size, w2size);
	}
}

double relu(double x)
{
	return x*(x > 0);
}

double relugrad(double y)
{
	return y > 0;
}

static void forward(LeNet5Cuda* lenetCuda, FeatureCuda* featuresCuda)
{
	ConvolutionForward(featuresCuda->input, featuresCuda->layer1, lenetCuda->weight0_1, lenetCuda->bias0_1,
					   INPUT, LAYER1, LENGTH_FEATURE0, LENGTH_FEATURE0);
	SubsampForward(featuresCuda->layer1, featuresCuda->layer2, LAYER1, LENGTH_FEATURE1, LAYER2, LENGTH_FEATURE2);
	ConvolutionForward(featuresCuda->layer2, featuresCuda->layer3, lenetCuda->weight2_3, lenetCuda->bias2_3,
					LAYER2, LAYER3, LENGTH_FEATURE2, LENGTH_FEATURE2);
	SubsampForward(featuresCuda->layer3, featuresCuda->layer4, LAYER3, LENGTH_FEATURE3, LAYER4, LENGTH_FEATURE4);
	ConvolutionForward(featuresCuda->layer4, featuresCuda->layer5, lenetCuda->weight4_5, lenetCuda->bias4_5,
						LAYER4, LAYER5, LENGTH_FEATURE4, LENGTH_FEATURE4);
	DotProductForward(featuresCuda->layer5, featuresCuda->output, lenetCuda->weight5_6, LAYER5, OUTPUT, lenetCuda->bias5_6);
}

static void backward(LeNet5Cuda* lenetCuda, LeNet5Cuda* deltasCuda, FeatureCuda* featuresCuda, FeatureCuda* errorsCuda)
{
	DotProductBackward(featuresCuda->layer5, errorsCuda->layer5, errorsCuda->output, lenetCuda->weight5_6, deltasCuda->weight5_6, deltasCuda->bias5_6, LAYER5, OUTPUT);
	ConvolutionBackward(featuresCuda->layer4, errorsCuda->layer4, errorsCuda->layer5, lenetCuda->weight4_5, deltasCuda->weight4_5, deltasCuda->bias4_5,
						LAYER4, LAYER5, LENGTH_FEATURE4, LENGTH_FEATURE4);
	SubsampBackward(featuresCuda->layer3, errorsCuda->layer3, errorsCuda->layer4, LENGTH_FEATURE3, LAYER3, LENGTH_FEATURE4);
	ConvolutionBackward(featuresCuda->layer2, errorsCuda->layer2, errorsCuda->layer3, lenetCuda->weight2_3, deltasCuda->weight2_3, deltasCuda->bias2_3,
					LAYER2, LAYER3, LENGTH_FEATURE2, LENGTH_FEATURE2);
	SubsampBackward(featuresCuda->layer1, errorsCuda->layer1, errorsCuda->layer2, LENGTH_FEATURE1, LAYER1, LENGTH_FEATURE2);
	ConvolutionBackward(featuresCuda->input, errorsCuda->input, errorsCuda->layer1, lenetCuda->weight0_1, deltasCuda->weight0_1, deltasCuda->bias0_1,
					INPUT, LAYER1, LENGTH_FEATURE0, LENGTH_FEATURE0);
}

static inline void load_input(FeatureCuda *features, image input)
{
	double layer0[LENGTH_FEATURE0][LENGTH_FEATURE0] = { 0 };
	const long sz = sizeof(image) / sizeof(**input);
	double mean = 0, std = 0;
	FOREACH(j, sizeof(image) / sizeof(*input))
		FOREACH(k, sizeof(*input) / sizeof(**input))
	{
		mean += input[j][k];
		std += input[j][k] * input[j][k];
	}
	mean /= sz;
	std = sqrt(std / sz - mean*mean);
	FOREACH(j, sizeof(image) / sizeof(*input))
		FOREACH(k, sizeof(*input) / sizeof(**input))
	{
		layer0[j + PADDING][k + PADDING] = (input[j][k] - mean) / std;
	}
	CUDAMEMCPY_CHECK(layer0, features->input, LENGTH_FEATURE0 * LENGTH_FEATURE0 * sizeof(double), hipMemcpyHostToDevice);
}

static inline void softmax(double input[OUTPUT], double loss[OUTPUT], int label, int count)
{
	double inner = 0;
	for (int i = 0; i < count; ++i)
	{
		double res = 0;
		for (int j = 0; j < count; ++j)
		{
			res += exp(input[j] - input[i]);
		}
		loss[i] = 1. / res;
		inner -= loss[i] * loss[i];
	}
	inner += loss[label];
	for (int i = 0; i < count; ++i)
	{
		loss[i] *= (i == label) - loss[i] - inner;
	}
}

static void load_target(FeatureCuda *features, FeatureCuda *errors, int label)
{
	double output[OUTPUT];
	double error[OUTPUT] = { 0 };
	CUDAMEMCPY_CHECK(features->output, output, OUTPUT * sizeof(double), hipMemcpyDeviceToHost);
	softmax(output, error, label, OUTPUT);
	CUDAMEMCPY_CHECK(error, errors->output, OUTPUT * sizeof(double), hipMemcpyHostToDevice);
}

static uint8 get_result(FeatureCuda *features, uint8 count)
{
	double output[OUTPUT] = { 1 };
	CUDAMEMCPY_CHECK(features->output, output, OUTPUT * sizeof(double), hipMemcpyDeviceToHost);
	uint8 result = 0;
	double maxvalue = *output;
	for (uint8 i = 1; i < count; ++i)
	{
		if (output[i] > maxvalue)
		{
			maxvalue = output[i];
			result = i;
		}
	}
	return result;
}

static double f64rand()
{
	static int randbit = 0;
	if (!randbit)
	{
		srand((unsigned)time(0));
		for (int i = RAND_MAX; i; i >>= 1, ++randbit);
	}
	unsigned long long lvalue = 0x4000000000000000L;
	int i = 52 - randbit;
	for (; i > 0; i -= randbit)
		lvalue |= (unsigned long long)rand() << i;
	lvalue |= (unsigned long long)rand() >> -i;
	return *(double *)&lvalue - 3;
}

void TrainBatch(LeNet5 *lenet, image *inputs, uint8 *labels, int batchSize, LeNet5Cuda* lenetCuda, LeNet5Cuda* deltasCuda, FeatureCuda* featuresCuda, FeatureCuda* errorsCuda)
{
	double buffer[GETCOUNT(LeNet5)] = { 0 };
	int i = 0;

	CUDAMEMCPY_CHECK(lenet->weight0_1, lenetCuda->weight0_1, sizeof(lenet->weight0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight2_3, lenetCuda->weight2_3, sizeof(lenet->weight2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight4_5, lenetCuda->weight4_5, sizeof(lenet->weight4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight5_6, lenetCuda->weight5_6, sizeof(lenet->weight5_6), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias0_1, lenetCuda->bias0_1, sizeof(lenet->bias0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias2_3, lenetCuda->bias2_3, sizeof(lenet->bias2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias4_5, lenetCuda->bias4_5, sizeof(lenet->bias4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias5_6, lenetCuda->bias5_6, sizeof(lenet->bias5_6), hipMemcpyHostToDevice);

	for (i = 0; i < batchSize; ++i)
	{ // For each training image
		LeNet5	deltas = { 0 };

		hipMemset(errorsCuda->layer3, 0, LAYER3 * LENGTH_FEATURE3 * LENGTH_FEATURE3 * sizeof(double));

		load_input(featuresCuda, inputs[i]);
		forward(lenetCuda, featuresCuda); // Forward propagation
		load_target(featuresCuda, errorsCuda, labels[i]);
		backward(lenetCuda, deltasCuda, featuresCuda, errorsCuda); // Backpropagation

		CUDAMEMCPY_CHECK(deltasCuda->weight0_1, deltas.weight0_1, sizeof(deltas.weight0_1), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->bias0_1, deltas.bias0_1, sizeof(deltas.bias0_1), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->weight2_3, deltas.weight2_3, sizeof(deltas.weight2_3), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->bias2_3, deltas.bias2_3, sizeof(deltas.bias2_3), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->weight4_5, deltas.weight4_5, sizeof(deltas.weight4_5), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->bias4_5, deltas.bias4_5, sizeof(deltas.bias4_5), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->weight5_6, deltas.weight5_6, sizeof(deltas.weight5_6), hipMemcpyDeviceToHost);
		CUDAMEMCPY_CHECK(deltasCuda->bias5_6, deltas.bias5_6, sizeof(deltas.bias5_6), hipMemcpyDeviceToHost);
		// if time, cudafy!
		FOREACH(j, GETCOUNT(LeNet5))
				buffer[j] += ((double *)&deltas)[j];
	}
	LeNet5	deltas = { 0 };
	double k = ALPHA / batchSize;
	FOREACH(i, GETCOUNT(LeNet5))
		((double *)lenet)[i] += k * buffer[i];
}

uint8 Predict(image input,uint8 count, LeNet5Cuda* lenetCuda, FeatureCuda* featuresCuda)
{
	load_input(featuresCuda, input);
	forward(lenetCuda, featuresCuda);
	return get_result(featuresCuda, count);
}

void Initial(LeNet5 *lenet)
{
	for (double *pos = (double *)lenet->weight0_1; pos < (double *)lenet->bias0_1; *pos++ = f64rand());
	for (double *pos = (double *)lenet->weight0_1; pos < (double *)lenet->weight2_3; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (INPUT + LAYER1))));
	for (double *pos = (double *)lenet->weight2_3; pos < (double *)lenet->weight4_5; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (LAYER2 + LAYER3))));
	for (double *pos = (double *)lenet->weight4_5; pos < (double *)lenet->weight5_6; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (LAYER4 + LAYER5))));
	for (double *pos = (double *)lenet->weight5_6; pos < (double *)lenet->bias0_1; *pos++ *= sqrt(6.0 / (LAYER5 + OUTPUT)));
	for (int *pos = (int *)lenet->bias0_1; pos < (int *)(lenet + 1); *pos++ = 0);
}

void PrintResult(int confusion_matrix[OUTPUT][OUTPUT])
{
	// Print the confusion matrix
	printf("%15sPredicted label\n%10s", " ", " ");
	for (int col = 0; col < 10; col++)
		printf("%6d", col);
	printf("%10s\n", "Total");
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\nTrue label\n");
	int row_labels = 0;
	int total = 0;
	for (int row = 0; row < 10; row++) {
		row_labels = 0;
		printf("%10d", row);
		for (int col = 0; col < 10; col++) {
			printf("%6d", confusion_matrix[row][col]);
			row_labels += confusion_matrix[row][col];
		}
		printf("%10d\n", row_labels);
		total += row_labels;
	}
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n%67s = %10d\n", "Total number of input images tested", total);
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n");
}
