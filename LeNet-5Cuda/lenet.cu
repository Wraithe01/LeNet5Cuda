#include "hip/hip_runtime.h"
/*
The MIT License(MIT)
Copyright(c) 2016 Fan Wen Jie

Permission is hereby granted, free of charge, to any person obtaining a copy
of this softwareand associated documentation files(the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions :

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
// Source: https://github.com/fan-wenjie/LeNet-5

#include "lenet.h"
#include <hip/hip_runtime.h>
#include <>
#include <memory.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <cstdio>


#define CUDAMALLOC_CHECK(ptr, size)									 \
{																	 \
	if (hipMalloc((void**)&ptr, size) != hipSuccess) {		     \
		fprintf(stderr, "ERROR: hipMalloc for %s failed!\n", #ptr); \
		return -1;													 \
	}																 \
}																	 \

int CudaInit()
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ERROR: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		return -1;
	}
	return 0;
}

int CudaDeInit()
{
	hipError_t cudaStatus;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "ERROR: hipDeviceReset failed!");
		return -1;
	}
	return 0;
}

int LeNetCudaAlloc(LeNet5Cuda* lenet5)
{
	// Allocate GPU buffers for LeNet5 data.
	CUDAMALLOC_CHECK(lenet5->weight0_1, INPUT * LAYER1 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight2_3, LAYER2 * LAYER3 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight4_5, LAYER4 * LAYER5 * LENGTH_KERNEL * LENGTH_KERNEL * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->weight5_6, (LAYER5 * LENGTH_FEATURE5 * LENGTH_FEATURE5) * OUTPUT * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias0_1, LAYER1 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias2_3, LAYER3 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias4_5, LAYER5 * sizeof(double));
	CUDAMALLOC_CHECK(lenet5->bias5_6, OUTPUT * sizeof(double));

	return 0;
}
int LeNetCudaFree(LeNet5Cuda* lenet5)
{
	hipFree(lenet5->weight0_1);
	hipFree(lenet5->weight2_3);
	hipFree(lenet5->weight4_5);
	hipFree(lenet5->weight5_6);
	hipFree(lenet5->bias0_1);
	hipFree(lenet5->bias2_3);
	hipFree(lenet5->bias4_5);
	hipFree(lenet5->bias5_6);
	return 0;
}

int FeatureCudaAlloc(FeatureCuda* feature)
{
	CUDAMALLOC_CHECK(feature->input, INPUT * LENGTH_FEATURE0 * LENGTH_FEATURE0 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer1, LAYER1 * LENGTH_FEATURE1 * LENGTH_FEATURE1 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer2, LAYER2 * LENGTH_FEATURE2 * LENGTH_FEATURE2 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer3, LAYER3 * LENGTH_FEATURE3 * LENGTH_FEATURE3 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer4, LAYER4 * LENGTH_FEATURE4 * LENGTH_FEATURE4 * sizeof(double));
	CUDAMALLOC_CHECK(feature->layer5, LAYER5 * LENGTH_FEATURE5 * LENGTH_FEATURE5 * sizeof(double));
	CUDAMALLOC_CHECK(feature->output, OUTPUT*sizeof(double));

	return 0;
}

int FeatureCudaFree(FeatureCuda* feature)
{
	hipFree(feature->input);
	hipFree(feature->layer1);
	hipFree(feature->layer2);
	hipFree(feature->layer3);
	hipFree(feature->layer4);
	hipFree(feature->layer5);
	hipFree(feature->output);
	return 0;
}

#define CUDAMEMCPY_CHECK(src, dest, bytes, type)												 \
	if (hipMemcpy(dest, src, bytes, type) != hipSuccess)								 \
		fprintf(stderr, "ERROR: cudaMemCpy %s from %s to %s failed!\n", #type, #src, #dest); \


#define GETLENGTH(array) (sizeof(array)/sizeof(*(array)))

#define GETCOUNT(array)  (sizeof(array)/sizeof(double))

#define FOREACH(i,count) for (int i = 0; i < count; ++i)

#define CONVOLUTE_VALID(input,output,weight)											\
{																						\
	FOREACH(o0,GETLENGTH(output))														\
		FOREACH(o1,GETLENGTH(*(output)))												\
			FOREACH(w0,GETLENGTH(weight))												\
				FOREACH(w1,GETLENGTH(*(weight)))										\
					(output)[o0][o1] += (input)[o0 + w0][o1 + w1] * (weight)[w0][w1];	\
}

#define CONVOLUTE_FULL(input,output,weight)												\
{																						\
	FOREACH(i0,GETLENGTH(input))														\
		FOREACH(i1,GETLENGTH(*(input)))													\
			FOREACH(w0,GETLENGTH(weight))												\
				FOREACH(w1,GETLENGTH(*(weight)))										\
					(output)[i0 + w0][i1 + w1] += (input)[i0][i1] * (weight)[w0][w1];	\
}

__global__ void ConvoluteKernelValid(double* input, double* output, double* weight, const int inputFeatures, const int inputW, const int inputH)
{
	int outFeature = blockIdx.z;
	int outputFeatures = gridDim.z;
	int threadW = blockIdx.x * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.x;
	int threadH = blockIdx.y * (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1) + threadIdx.y;

	double acc = 0;

	__shared__ double inTile[LENGTH_KERNEL_TILE][LENGTH_KERNEL_TILE];
	__shared__ double weightFilter[LENGTH_KERNEL][LENGTH_KERNEL];
	
	for (int inFeature = 0; inFeature < inputFeatures; inFeature++)
	{
		if ((threadW < inputW) && (threadH < inputH))
		{
			inTile[threadIdx.y][threadIdx.x] = input[inFeature * inputH * inputW +
				threadH * inputW +
				threadW];
		}
		if ((threadIdx.x < LENGTH_KERNEL) && (threadIdx.y < LENGTH_KERNEL))
		{
			weightFilter[threadIdx.y][threadIdx.x] = weight[inFeature * gridDim.z * LENGTH_KERNEL * LENGTH_KERNEL +
				outFeature * LENGTH_KERNEL * LENGTH_KERNEL +
				threadIdx.y * LENGTH_KERNEL +
				threadIdx.x];
		}
		__syncthreads();
		if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
			(threadH < (inputH - LENGTH_KERNEL + 1)) && (threadW < (inputW - LENGTH_KERNEL + 1)))
		{
			for (int p = 0; p < LENGTH_KERNEL; p++)
			{
				for (int q = 0; q < LENGTH_KERNEL; q++)
				{
					acc += inTile[threadIdx.y + p][threadIdx.x + q] * weightFilter[p][q];
				}
			}
		}
	}

	if ((threadIdx.x < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) && (threadIdx.y < (LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1)) &&
		(threadH < (inputH - LENGTH_KERNEL + 1)) && (threadW < (inputW - LENGTH_KERNEL + 1)))
	{
		output[outFeature * (inputH - LENGTH_KERNEL + 1) * (inputW - LENGTH_KERNEL + 1) + threadH * (inputW - LENGTH_KERNEL + 1) + threadW] = acc;
	}
}

void ConvoluteValid(double* input, double* output, double* weight, const int inputFeatures, const int outputFeatures, const int inputWidth, const int inputHeight)
{
	/*Blocks are fixed sized tiles to allow for any size of input*/
	dim3 block(LENGTH_KERNEL_TILE, LENGTH_KERNEL_TILE, 1);
	unsigned int tilesW = ceil((float)(inputWidth - LENGTH_KERNEL + 1) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	unsigned int tilesH = ceil((float)(inputHeight - LENGTH_KERNEL + 1) / (float)(LENGTH_KERNEL_TILE - LENGTH_KERNEL + 1));
	dim3 grid(tilesW, tilesH, outputFeatures);
	ConvoluteKernelValid <<< grid, block >>> (input, output, weight, inputFeatures, inputWidth, inputHeight);
}

__global__ void ForwardReluKernel(double* feature, double* bias, const int featureWidth, const int featureHeight)
{
	int width = blockIdx.x * LENGTH_KERNEL_TILE + threadIdx.x;
	int height = blockIdx.y * LENGTH_KERNEL_TILE + threadIdx.y;
	int featureMap = blockIdx.z;

	if ((width < featureWidth) && (height < featureHeight))
	{
		if ((feature[featureMap * featureHeight * featureWidth + height * featureWidth + width] + bias[featureMap]) < 0)
			feature[featureMap * featureHeight * featureWidth + height * featureWidth + width] = 0;
	}
}

void ForwardRelu(double* feature, double* bias, const int featureCount, const int featureWidth, const int featureHeight)
{
	/*Blocks are fixed sized tiles to allow for any size of input*/
	dim3 block(LENGTH_KERNEL_TILE, LENGTH_KERNEL_TILE, 1);
	unsigned int tilesW = ceil((float)featureWidth / (float)LENGTH_KERNEL_TILE);
	unsigned int tilesH = ceil((float)featureHeight / (float)LENGTH_KERNEL_TILE);
	dim3 grid(tilesW, tilesH, featureCount);
	ForwardReluKernel <<< grid, block >>> (feature, bias, featureWidth, featureHeight);
}

// Similar functionality as the code in Figure 16.4 of the textbook
void ConvolutionForward(double* input, double* output, double* weight, double* bias, const int inputFeatures, const int outputFeatures, const int inputWidth, const int inputHeight)					\
{
	ConvoluteValid(input, output, weight, inputFeatures, outputFeatures, inputWidth, inputHeight);
	ForwardRelu(output, bias, outputFeatures, inputWidth - LENGTH_KERNEL + 1, inputHeight - LENGTH_KERNEL + 1);
}

#define CONVOLUTION_BACKWARD(input,inerror,outerror,weight,wd,bd,actiongrad)\
{																			\
	for (int x = 0; x < GETLENGTH(weight); ++x)								\
		for (int y = 0; y < GETLENGTH(*weight); ++y)						\
			CONVOLUTE_FULL(outerror[y], inerror[x], weight[x][y]);			\
	FOREACH(i, GETCOUNT(inerror))											\
		((double *)inerror)[i] *= actiongrad(((double *)input)[i]);			\
	FOREACH(j, GETLENGTH(outerror))											\
		FOREACH(i, GETCOUNT(outerror[j]))									\
		bd[j] += ((double *)outerror[j])[i];								\
	for (int x = 0; x < GETLENGTH(weight); ++x)								\
		for (int y = 0; y < GETLENGTH(*weight); ++y)						\
			CONVOLUTE_VALID(input[x], wd[x][y], outerror[y]);				\
}

// Similar functionality as the code in Figure 16.5 of the textbook
#define SUBSAMP_MAX_FORWARD(input,output)														\
{																								\
	const int len0 = GETLENGTH(*(input)) / GETLENGTH(*(output));								\
	const int len1 = GETLENGTH(**(input)) / GETLENGTH(**(output));								\
	FOREACH(i, GETLENGTH(output))																\
	FOREACH(o0, GETLENGTH(*(output)))															\
	FOREACH(o1, GETLENGTH(**(output)))															\
	{																							\
		int x0 = 0, x1 = 0, ismax;																\
		FOREACH(l0, len0)																		\
			FOREACH(l1, len1)																	\
		{																						\
			ismax = input[i][o0*len0 + l0][o1*len1 + l1] > input[i][o0*len0 + x0][o1*len1 + x1];\
			x0 += ismax * (l0 - x0);															\
			x1 += ismax * (l1 - x1);															\
		}																						\
		output[i][o0][o1] = input[i][o0*len0 + x0][o1*len1 + x1];								\
	}																							\
}

#define SUBSAMP_MAX_BACKWARD(input,inerror,outerror)											\
{																								\
	const int len0 = GETLENGTH(*(inerror)) / GETLENGTH(*(outerror));							\
	const int len1 = GETLENGTH(**(inerror)) / GETLENGTH(**(outerror));							\
	FOREACH(i, GETLENGTH(outerror))																\
	FOREACH(o0, GETLENGTH(*(outerror)))															\
	FOREACH(o1, GETLENGTH(**(outerror)))														\
	{																							\
		int x0 = 0, x1 = 0, ismax;																\
		FOREACH(l0, len0)																		\
			FOREACH(l1, len1)																	\
		{																						\
			ismax = input[i][o0*len0 + l0][o1*len1 + l1] > input[i][o0*len0 + x0][o1*len1 + x1];\
			x0 += ismax * (l0 - x0);															\
			x1 += ismax * (l1 - x1);															\
		}																						\
		inerror[i][o0*len0 + x0][o1*len1 + x1] = outerror[i][o0][o1];							\
	}																							\
}

#define DOT_PRODUCT_FORWARD(input,output,weight,bias,action)				\
{																			\
	for (int x = 0; x < GETLENGTH(weight); ++x)								\
		for (int y = 0; y < GETLENGTH(*weight); ++y)						\
			((double *)output)[y] += ((double *)input)[x] * weight[x][y];	\
	FOREACH(j, GETLENGTH(bias))												\
		((double *)output)[j] = action(((double *)output)[j] + bias[j]);	\
}

#define DOT_PRODUCT_BACKWARD(input,inerror,outerror,weight,wd,bd,actiongrad)	\
{																				\
	for (int x = 0; x < GETLENGTH(weight); ++x)									\
		for (int y = 0; y < GETLENGTH(*weight); ++y)							\
			((double *)inerror)[x] += ((double *)outerror)[y] * weight[x][y];	\
	FOREACH(i, GETCOUNT(inerror))												\
		((double *)inerror)[i] *= actiongrad(((double *)input)[i]);				\
	FOREACH(j, GETLENGTH(outerror))												\
		bd[j] += ((double *)outerror)[j];										\
	for (int x = 0; x < GETLENGTH(weight); ++x)									\
		for (int y = 0; y < GETLENGTH(*weight); ++y)							\
			wd[x][y] += ((double *)input)[x] * ((double *)outerror)[y];			\
}

double relu(double x)
{
	return x*(x > 0);
}

double relugrad(double y)
{
	return y > 0;
}

static void forward(LeNet5 *lenet, Feature *features, double(*action)(double), LeNet5Cuda* lenetCuda, FeatureCuda* featuresCuda)
{
	CUDAMEMCPY_CHECK(lenet->weight0_1, lenetCuda->weight0_1, sizeof(lenet->weight0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight2_3, lenetCuda->weight2_3, sizeof(lenet->weight2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight4_5, lenetCuda->weight4_5, sizeof(lenet->weight4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->weight5_6, lenetCuda->weight5_6, sizeof(lenet->weight5_6), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias0_1, lenetCuda->bias0_1, sizeof(lenet->bias0_1), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias2_3, lenetCuda->bias2_3, sizeof(lenet->bias2_3), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias4_5, lenetCuda->bias4_5, sizeof(lenet->bias4_5), hipMemcpyHostToDevice);
	CUDAMEMCPY_CHECK(lenet->bias5_6, lenetCuda->bias5_6, sizeof(lenet->bias5_6), hipMemcpyHostToDevice);

	CUDAMEMCPY_CHECK(features->input, featuresCuda->input, sizeof(features->input), hipMemcpyHostToDevice);
	ConvolutionForward(featuresCuda->input, featuresCuda->layer1, lenetCuda->weight0_1, lenetCuda->bias0_1,
					   INPUT, LAYER1, LENGTH_FEATURE0, LENGTH_FEATURE0);
	
	CUDAMEMCPY_CHECK(featuresCuda->layer1, features->layer1, sizeof(features->layer1), hipMemcpyDeviceToHost);
	SUBSAMP_MAX_FORWARD(features->layer1, features->layer2);
	
	CUDAMEMCPY_CHECK(features->layer2, featuresCuda->layer2, sizeof(features->layer2), hipMemcpyHostToDevice);
	ConvolutionForward(featuresCuda->layer2, featuresCuda->layer3, lenetCuda->weight2_3, lenetCuda->bias2_3,
					LAYER2, LAYER3, LENGTH_FEATURE2, LENGTH_FEATURE2);
	
	CUDAMEMCPY_CHECK(featuresCuda->layer3, features->layer3, sizeof(features->layer3), hipMemcpyDeviceToHost);
	SUBSAMP_MAX_FORWARD(features->layer3, features->layer4);
	
	CUDAMEMCPY_CHECK(features->layer4, featuresCuda->layer4, sizeof(features->layer4), hipMemcpyHostToDevice);
	ConvolutionForward(featuresCuda->layer4, featuresCuda->layer5, lenetCuda->weight4_5, lenetCuda->bias4_5,
						LAYER4, LAYER5, LENGTH_FEATURE4, LENGTH_FEATURE4);
	
	CUDAMEMCPY_CHECK(featuresCuda->layer5, features->layer5, sizeof(features->layer5), hipMemcpyDeviceToHost);
	DOT_PRODUCT_FORWARD(features->layer5, features->output, lenet->weight5_6, lenet->bias5_6, action);
}

static void backward(LeNet5 *lenet, LeNet5 *deltas, Feature *errors, Feature *features, double(*actiongrad)(double))
{
	DOT_PRODUCT_BACKWARD(features->layer5, errors->layer5, errors->output, lenet->weight5_6, deltas->weight5_6, deltas->bias5_6, actiongrad);
	CONVOLUTION_BACKWARD(features->layer4, errors->layer4, errors->layer5, lenet->weight4_5, deltas->weight4_5, deltas->bias4_5, actiongrad);
	SUBSAMP_MAX_BACKWARD(features->layer3, errors->layer3, errors->layer4);
	CONVOLUTION_BACKWARD(features->layer2, errors->layer2, errors->layer3, lenet->weight2_3, deltas->weight2_3, deltas->bias2_3, actiongrad);
	SUBSAMP_MAX_BACKWARD(features->layer1, errors->layer1, errors->layer2);
	CONVOLUTION_BACKWARD(features->input, errors->input, errors->layer1, lenet->weight0_1, deltas->weight0_1, deltas->bias0_1, actiongrad);
}

static inline void load_input(Feature *features, image input)
{
	double (*layer0)[LENGTH_FEATURE0][LENGTH_FEATURE0] = features->input;
	const long sz = sizeof(image) / sizeof(**input);
	double mean = 0, std = 0;
	FOREACH(j, sizeof(image) / sizeof(*input))
		FOREACH(k, sizeof(*input) / sizeof(**input))
	{
		mean += input[j][k];
		std += input[j][k] * input[j][k];
	}
	mean /= sz;
	std = sqrt(std / sz - mean*mean);
	FOREACH(j, sizeof(image) / sizeof(*input))
		FOREACH(k, sizeof(*input) / sizeof(**input))
	{
		layer0[0][j + PADDING][k + PADDING] = (input[j][k] - mean) / std;
	}
}

static inline void softmax(double input[OUTPUT], double loss[OUTPUT], int label, int count)
{
	double inner = 0;
	for (int i = 0; i < count; ++i)
	{
		double res = 0;
		for (int j = 0; j < count; ++j)
		{
			res += exp(input[j] - input[i]);
		}
		loss[i] = 1. / res;
		inner -= loss[i] * loss[i];
	}
	inner += loss[label];
	for (int i = 0; i < count; ++i)
	{
		loss[i] *= (i == label) - loss[i] - inner;
	}
}

static void load_target(Feature *features, Feature *errors, int label)
{
	double *output = (double *)features->output;
	double *error = (double *)errors->output;
	softmax(output, error, label, GETCOUNT(features->output));
}

static uint8 get_result(Feature *features, uint8 count)
{
	double *output = (double *)features->output; 
	const int outlen = GETCOUNT(features->output);
	uint8 result = 0;
	double maxvalue = *output;
	for (uint8 i = 1; i < count; ++i)
	{
		if (output[i] > maxvalue)
		{
			maxvalue = output[i];
			result = i;
		}
	}
	return result;
}

static double f64rand()
{
	static int randbit = 0;
	if (!randbit)
	{
		//srand((unsigned)time(0));
		srand(0);
		for (int i = RAND_MAX; i; i >>= 1, ++randbit);
	}
	unsigned long long lvalue = 0x4000000000000000L;
	int i = 52 - randbit;
	for (; i > 0; i -= randbit)
		lvalue |= (unsigned long long)rand() << i;
	lvalue |= (unsigned long long)rand() >> -i;
	return *(double *)&lvalue - 3;
}

void TrainBatch(LeNet5 *lenet, image *inputs, uint8 *labels, int batchSize, LeNet5Cuda* lenetCuda, LeNet5Cuda* deltasCuda, FeatureCuda* featuresCuda, FeatureCuda* errorsCuda)
{
	double buffer[GETCOUNT(LeNet5)] = { 0 };
	int i = 0;
	for (i = 0; i < batchSize; ++i)
	{ // For each training image
		// should be able to delete these once all parts are moved to cuda
		Feature features = { 0 };
		Feature errors = { 0 };
		LeNet5	deltas = { 0 };

		load_input(&features, inputs[i]);
		forward(lenet, &features, relu, lenetCuda, featuresCuda); // Forward propagation
		load_target(&features, &errors, labels[i]);
		/*
		CUDAMEMCPY_CHECK(deltas.weight0_1, deltasCuda->weight0_1, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.weight2_3, deltasCuda->weight2_3, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.weight4_5, deltasCuda->weight4_5, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.weight5_6, deltasCuda->weight5_6, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.bias0_1, deltasCuda->bias0_1, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.bias2_3, deltasCuda->bias2_3, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.bias4_5, deltasCuda->bias4_5, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(deltas.bias5_6, deltasCuda->bias5_6, hipMemcpyHostToDevice);

		CUDAMEMCPY_CHECK(errors.input, errorsCuda->input, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.layer1, errorsCuda->layer1, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.layer2, errorsCuda->layer2, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.layer3, errorsCuda->layer3, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.layer4, errorsCuda->layer4, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.layer5, errorsCuda->layer5, hipMemcpyHostToDevice);
		CUDAMEMCPY_CHECK(errors.output, errorsCuda->output, hipMemcpyHostToDevice);
		*/

		backward(lenet, &deltas, &errors, &features, relugrad); // Backpropagation
		FOREACH(j, GETCOUNT(LeNet5))
				buffer[j] += ((double *)&deltas)[j];
	}
	double k = ALPHA / batchSize;
	FOREACH(i, GETCOUNT(LeNet5))
		((double *)lenet)[i] += k * buffer[i];
}

/*
void Train(LeNet5 *lenet, image input, uint8 label)
{
	Feature features = { 0 };
	Feature errors = { 0 };
	LeNet5 deltas = { 0 };
	load_input(&features, input);
	forward(lenet, &features, relu);
	load_target(&features, &errors, label);
	backward(lenet, &deltas, &errors, &features, relugrad);
	FOREACH(i, GETCOUNT(LeNet5))
		((double *)lenet)[i] += ALPHA * ((double *)&deltas)[i];
}
*/

uint8 Predict(LeNet5 *lenet, image input,uint8 count, LeNet5Cuda* lenetCuda, FeatureCuda* featuresCuda)
{
	Feature features = { 0 };
	load_input(&features, input);
	forward(lenet, &features, relu, lenetCuda, featuresCuda);
	return get_result(&features, count);
}

void Initial(LeNet5 *lenet)
{
	for (double *pos = (double *)lenet->weight0_1; pos < (double *)lenet->bias0_1; *pos++ = f64rand());
	for (double *pos = (double *)lenet->weight0_1; pos < (double *)lenet->weight2_3; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (INPUT + LAYER1))));
	for (double *pos = (double *)lenet->weight2_3; pos < (double *)lenet->weight4_5; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (LAYER2 + LAYER3))));
	for (double *pos = (double *)lenet->weight4_5; pos < (double *)lenet->weight5_6; *pos++ *= sqrt(6.0 / (LENGTH_KERNEL * LENGTH_KERNEL * (LAYER4 + LAYER5))));
	for (double *pos = (double *)lenet->weight5_6; pos < (double *)lenet->bias0_1; *pos++ *= sqrt(6.0 / (LAYER5 + OUTPUT)));
	for (int *pos = (int *)lenet->bias0_1; pos < (int *)(lenet + 1); *pos++ = 0);
}

void PrintResult(int confusion_matrix[OUTPUT][OUTPUT])
{
	// Print the confusion matrix
	printf("%15sPredicted label\n%10s", " ", " ");
	for (int col = 0; col < 10; col++)
		printf("%6d", col);
	printf("%10s\n", "Total");
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\nTrue label\n");
	int row_labels = 0;
	int total = 0;
	for (int row = 0; row < 10; row++) {
		row_labels = 0;
		printf("%10d", row);
		for (int col = 0; col < 10; col++) {
			printf("%6d", confusion_matrix[row][col]);
			row_labels += confusion_matrix[row][col];
		}
		printf("%10d\n", row_labels);
		total += row_labels;
	}
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n%67s = %10d\n", "Total number of input images tested", total);
	for (int n = 0; n < 70; n++)
		printf("%s", "-");
	printf("\n");
}
